// Copyright 2004-, Facebook, Inc. All Rights Reserved.

#include <stdio.h>
#include <stdexcept>
#include <hip/hip_runtime.h>

#include "HalfPrec.h"
#include "util/Transform.cuh"

using namespace facebook::CUDAUtil;
void halfprec_ToHalf(hipStream_t stream,
                     const float* input,
                     half_t* output,
                     size_t n) {
  transform<ToHalf>(stream, input, output, n);
}

void halfprec_ToFloat(hipStream_t stream,
                      const half_t* input,
                      float* output,
                      size_t n) {
  transform<ToFloat>(stream, input, output, n);
}
