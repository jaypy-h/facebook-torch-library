// Copyright 2004-present Facebook. All Rights Reserved.

#pragma once

#include "cuda/fbfft/FBFFT.h"
#include "cuda/fbfft/FBFFTCommon.cuh"

namespace facebook { namespace cuda { namespace fbfft {

template
facebook::cuda::fbfft::FBFFTParameters::ErrorCode fbfft1D<1>(
    DeviceTensor<float, 2>& real,
    DeviceTensor<float, 3>& complex,
    hipStream_t s);

template
facebook::cuda::fbfft::FBFFTParameters::ErrorCode fbfft2D<1>(
    DeviceTensor<float, 3>& real,
    DeviceTensor<float, 4>& complex,
    hipStream_t s);

template
facebook::cuda::fbfft::FBFFTParameters::ErrorCode fbfft2D<1>(
    DeviceTensor<Complex, 3>& complexSrc,
    DeviceTensor<Complex, 3>& complexDst,
    hipStream_t s);

template
facebook::cuda::fbfft::FBFFTParameters::ErrorCode fbifft1D<1>(
    DeviceTensor<float, 2>& real,
    DeviceTensor<float, 3>& complex,
    hipStream_t s);

template
facebook::cuda::fbfft::FBFFTParameters::ErrorCode fbifft2D<1>(
    DeviceTensor<float, 4>& srcComplexAsFloat,
    DeviceTensor<float, 4>& dstComplexAsFloat,
    hipStream_t s);

template
facebook::cuda::fbfft::FBFFTParameters::ErrorCode fbifft2D<1>(
    DeviceTensor<Complex, 3>& srcComplex,
    DeviceTensor<float, 3>& realDst,
    hipStream_t s);

}}}
