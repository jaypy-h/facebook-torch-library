#include "hip/hip_runtime.h"
// Copyright 2004-present Facebook. All Rights Reserved.

#include "CuFFTConvolution_UpdateOutput.cuh"

#include "cuda/CudaUtils.cuh"
#include "cuda/DeviceTensor.cuh"
#include "DeviceTensorUtils.h"
#include "THCTensor.h"
#include "ConvolutionBias.cuh"
#include "CuBLASWrapper.h"
#include "CuFFTWrapper.cuh"
#include "CuFFTConvolution.cuh"
#include "Utils.cuh"

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <glog/logging.h>

using namespace facebook::cuda;

namespace facebook { namespace deeplearning { namespace torch {

// Assumes complex is float[2]
__global__ void referenceUpdateOuput(DeviceTensor<float, 5> inputComplex,
                                     DeviceTensor<float, 5> filtersComplex,
                                     DeviceTensor<float, 5> outputComplex)
{
  // Input originally real, we have circular Hermitian symmetry:
  // X[k] = X∗[−k mod N] .
  const int Batches = inputComplex.getSize(0);
  const int Filters = filtersComplex.getSize(0);
  const int OutputRows = outputComplex.getSize(2);
  const int OutputCols = outputComplex.getSize(3);
  for (int batch = 0; batch < Batches; ++batch) {
    for (int filter = 0; filter < Filters; ++filter) {
      for (int outputRow = 0; outputRow < OutputRows; ++outputRow) {
        for (int outputCol = 0; outputCol < OutputCols; ++outputCol) {
          hipFloatComplex* out = outputComplex[batch][filter]
            [outputRow][outputCol].dataAs<hipFloatComplex>();
          out->x = 0.0f;
          out->y = 0.0f;
          for (int inputPlane = 0; inputPlane < inputComplex.getSize(1);
               ++inputPlane) {
            hipFloatComplex input =
              inputComplex[batch][inputPlane]
              [outputRow][outputCol].ldgAs<hipFloatComplex>();

            hipFloatComplex filters =
              hipConjf(filtersComplex[filter][inputPlane]
                      [outputRow][outputCol].ldgAs<hipFloatComplex>());

            *out = hipCfmaf(input, filters, *out);
          }
        }
      }
    }
  }
}

void CuFFTConvolution_ReferenceUpdateOutput(THCState* state,
                                            THCudaTensor* inputTH,
                                            THCudaTensor* kernelsTH,
                                            THCudaTensor* outputTH,
                                            THCudaTensor* biasTH,
                                            THCudaTensor* inputComplexTH,
                                            THCudaTensor* kernelsComplexTH,
                                            THCudaTensor* outputComplexTH) {
  DeviceTensor<float, 4> filters =
    torchToDeviceTensor<float, 4>(state, kernelsTH);
  DeviceTensor<float, 4> input =
    torchToDeviceTensor<float, 4>(state, inputTH);
  DeviceTensor<float, 4> output =
    torchToDeviceTensor<float, 4>(state, outputTH);

  DeviceTensor<float, 5> inputComplex =
    torchToDeviceTensor<float, 5>(state, inputComplexTH);
  DeviceTensor<float, 5> outputComplex =
    torchToDeviceTensor<float, 5>(state, outputComplexTH);
  DeviceTensor<float, 5> filtersComplex =
    torchToDeviceTensor<float, 5>(state, kernelsComplexTH);

  fft2d<2>(input, inputComplex);
  fft2d<2>(filters, filtersComplex);

  dim3 grid(1);
  dim3 block(1);
  referenceUpdateOuput<<<grid, block>>>(
    inputComplex, filtersComplex, outputComplex);

  fft2d<2>(output, outputComplex, FFTParameters().inverse());

  bias::updateOutputBias(state, outputTH, biasTH);
}

void CuFFTConvolution_UpdateOutput(THCState* state,
                                   THCudaTensor* inputTH,
                                   THCudaTensor* kernelsTH,
                                   THCudaTensor* outputTH,
                                   THCudaTensor* biasTH,
                                   THCudaTensor* inputComplexTH,
                                   THCudaTensor* kernelsComplexTH,
                                   THCudaTensor* outputComplexTH,
                                   THCudaTensor* inputComplexTTH,
                                   THCudaTensor* kernelsComplexTTH,
                                   THCudaTensor* outputComplexTTH) {
  CuFFTConvolution conv((ConvolutionPass(ConvolutionPass::kUpdateOutput)));
  conv.withInputAndBuffers(state,
                           inputTH, inputComplexTH, inputComplexTTH)
    .withFiltersAndBuffers(state,
                           kernelsTH, kernelsComplexTH, kernelsComplexTTH)
    .withOutputAndBuffers(state,
                          outputTH, outputComplexTH, outputComplexTTH)
    .run();

  bias::updateOutputBias(state, outputTH, biasTH);
}

void CuFFTConvolution_UpdateOutput(THCState* state,
                                   CuFFTConvolution* conv,
                                   THCudaTensor* outputTH,
                                   THCudaTensor* biasTH) {
  conv->run();

  bias::updateOutputBias(state, outputTH, biasTH);
}

} } } // namespace
