#include "hip/hip_runtime.h"
// Copyright 2004-present Facebook. All Rights Reserved.
#include "cuda/WarpReductionsTestBindings.cuh"
#include "cuda/WarpReductions.cuh"

#include <hip/hip_runtime.h>
#include <stdio.h>

using namespace std;

namespace facebook { namespace cuda {

__device__ int hasDuplicate[32];

__global__ void checkDuplicates(int num, int* v) {
  hasDuplicate[threadIdx.x] = (int) warpHasCollision(v[threadIdx.x]);
}

__device__ unsigned int duplicateMask;

__global__ void checkDuplicateMask(int num, int* v) {
  unsigned int mask = warpCollisionMask(v[threadIdx.x]);
  if (threadIdx.x == 0) {
    duplicateMask = mask;
  }
}

vector<int> hostCheckDuplicates(const vector<int>& v) {
  int* devSet = NULL;
  hipMalloc(&devSet, v.size() * sizeof(int));
  hipMemcpy(devSet, v.data(), v.size() * sizeof(int),
             hipMemcpyHostToDevice);

  checkDuplicates<<<1, 32>>>(v.size(), devSet);

  vector<int> hasDuplicates(32, false);
  hipMemcpyFromSymbol(hasDuplicates.data(),
                       hasDuplicate, sizeof(int) * 32, 0,
                       hipMemcpyDeviceToHost);
  hipFree(devSet);

  return hasDuplicates;
}

unsigned int hostCheckDuplicateMask(const vector<int>& v) {
  int* devSet = NULL;
  hipMalloc(&devSet, v.size() * sizeof(int));
  hipMemcpy(devSet, v.data(), v.size() * sizeof(int),
             hipMemcpyHostToDevice);

  checkDuplicateMask<<<1, 32>>>(v.size(), devSet);

  unsigned int mask = 0;
  hipMemcpyFromSymbol(&mask,
                       HIP_SYMBOL(duplicateMask), sizeof(unsigned int), 0,
                       hipMemcpyDeviceToHost);
  hipFree(devSet);

  return mask;
}

} }
